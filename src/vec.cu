#include "hip/hip_runtime.h"
//
//  vec.cpp
//  CUDA Physics
//
//  Created by Jacob Austin on 5/13/18.
//  Copyright © 2018 Jacob Austin. All rights reserved.
//

#include "vec.h"


// https://en.wikipedia.org/wiki/Rodrigues%27_rotation_formula
// rotate a vector {v_} with rotation axis {k} anchored at point {offset} by {theta} [rad]
CUDA_CALLABLE_MEMBER Vec AxisAngleRotaion(const Vec& k, const Vec& v_, const double& theta, const Vec& offset) {
	Vec v = v_ - offset;
	double c = cos(theta);
	Vec v_rot = v * c + cross(k, v) * sin(theta) + dot(k,v) * (1 - c) * k;
	//Vec v_rot = cross(k, v) * sin(theta);
	//v_rot += v * c;
	//v_rot += dot(k, v) * (1 - c) * k;
	v_rot += offset;
	return v_rot;
}