#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif
#ifdef __HIPCC__
#define CUDA_DEVICE __device__
#else
#define CUDA_DEVICE
#endif

#include<hip/hip_runtime.h>
#include<cuda_device_runtime_api.h>
#include<>

#include <thrust/device_vector.h>
#include<thrust/system/cuda/experimental/pinned_allocator.h>

#include <chrono> // for time measurement

#include "shader.h"
#include "object.h"
#include "sim.h"


#define _USE_MATH_DEFINES
#include <math.h>

#include <omp.h>

#include <msgpack.hpp>

#include <thread>

#include "vec.h"
#include <complex>


//template<class T> // alias template for pinned allocator
//using ThurstHostVec = std::vector<T, thrust::system::cuda::experimental::pinned_allocator<T>>;


int main()
{
	auto start = std::chrono::steady_clock::now();

	const int num_body = 5;//number of bodies
	Model bot("..\\src\\data.msgpack"); //defined in sim.h

	int num_mass = bot.vertices.size(); // number of mass
	int num_spring = bot.edges.size(); // number of spring

	Simulation sim(num_mass, num_spring); // Simulation object
	MASS& mass = sim.mass; // reference variable for sim.mass
	SPRING& spring = sim.spring; // reference variable for sim.spring

	sim.global_acc = Vec(0, 0, -9.8); // global acceleration
	sim.dt = 4e-5; // timestep

	double m = 8e-4;// mass per vertex

	double spring_constant = 6e2; //spring constant for silicone leg
	double spring_damping = 0.4; // damping for spring

	double scale_high = 2.5;// scaling factor
	double scale_low = 0.2;

	double spring_constant_high = spring_constant* scale_high;//spring constant for rigid spring
	double spring_constant_low = spring_constant* scale_low;// spring constant for resetable spring

	printf("total mass:%.2f kg\n", m * num_mass);

#pragma omp parallel for
	for (int i = 0; i < num_mass; i++)
	{
		mass.pos[i]= bot.vertices[i]; // position (Vec) [m]
		mass.color[i]= bot.colors[i]; // color (Vec) [0.0-1.0]
		mass.m[i] = m; // mass [kg]
		mass.constrain[i] = bot.isSurface[i];// set constrain to true for suface points, and false otherwise
	}
#pragma omp parallel for
	for (int i = 0; i < num_spring; i++)
	{
		spring.left[i] = bot.edges[i][0]; // the left mass index of the spring
		spring.right[i] = bot.edges[i][1]; // the right mass index of the spring
		spring.k[i] = spring_constant; // spring constant
		spring.damping[i] = spring_damping; // spring constant
		spring.rest[i] = (mass.pos[spring.left[i]] - mass.pos[spring.right[i]]).norm(); // spring rest length
		spring.resetable[i] = false; // set all spring as non-resetable
	}

	// bot.idVertices: body,leg0,leg1,leg2,leg3,anchor0,anchor1,anchor2,anchor3,\
					oxyz_body,oxyz_joint0_body,oxyz_joint0_leg0,oxyz_joint1_body,oxyz_joint1_leg1,\
							  oxyz_joint2_body,oxyz_joint2_leg2,oxyz_joint3_body,oxyz_joint3_leg3,the end
	// bot.idEdges: body, leg0, leg1, leg2, leg3, anchors, rotsprings, fricsprings, oxyz_self_springs, oxyz_anchor_springs, the end

	// set higher spring constant for the robot body
	for (int i = 0; i < bot.idEdges[1]; i++)
	{
		spring.k[i] = spring_constant_high;
	}
	// set higher spring constant for the rotational joints
	for (int i = bot.idEdges[num_body]; i < bot.idEdges[num_body +1]; i++)
	{
		spring.k[i] = spring_constant_high; // joints anchors
	}
	for (int i = bot.idEdges[num_body +1]; i < bot.idEdges[num_body +2]; i++)
	{
		spring.k[i] = spring_constant_high; // joints rotation spring
	}

	sim.id_restable_spring_start = bot.idEdges[num_body + 2]; // resetable spring (frictional spring)
	sim.id_resetable_spring_end = bot.idEdges[num_body + 3];
	for (int i = sim.id_restable_spring_start; i < sim.id_resetable_spring_end; i++)
	{
		spring.k[i] = spring_constant_low;// resetable spring, reset the rest length per dynamic update
		spring.damping[i] = spring_damping*1.5;
		spring.resetable[i] = true;
	}


	

	//// the start (inclusive) and end (exclusive) index of the anchor points
	//double id_joint_anchor_start = bot.idVertices[num_body];
	//double id_joint_anchor_end = bot.idVertices[num_body + sim.num_joint];

	//oxyz_body,oxyz_joint0_body,oxyz_joint0_leg0,oxyz_joint1_body,oxyz_joint1_leg1,\
				oxyz_joint2_body,oxyz_joint2_leg2,oxyz_joint3_body,oxyz_joint3_leg3,
	sim.id_oxyz_start = bot.idVertices[num_body + sim.num_joint];
	sim.id_oxyz_end = bot.idVertices[num_body + sim.num_joint + 1 + 2* sim.num_joint];

	double scale_down = 0.1;
	// set lower mass for the anchored coordinate systems
	for (int i = sim.id_oxyz_start; i < sim.id_oxyz_end; i++)
	{
		mass.m[i] = m * scale_down; // mass [kg]
	}

	for (int i = bot.idEdges[num_body + 3]; i < bot.idEdges[num_body + 4]; i++)
	{
		spring.k[i] = spring_constant * 2*scale_down;// oxyz_self_springs
		spring.damping[i] = spring_damping * scale_down;
	}
	for (int i = bot.idEdges[num_body + 4]; i < bot.idEdges[num_body + 5]; i++)
	{
		spring.k[i] = spring_constant * scale_down;// oxyz_anchor_springs
		spring.damping[i] = spring_damping * scale_down;
	}


	sim.joints.init(bot.Joints, true);
	sim.d_joints.init(bot.Joints, false);
	sim.d_joints.copyFrom(sim.joints);


	// set max speed for each joint
	sim.max_joint_speed = 200. / 60. * 2 * 3.1415926 * sim.dt;//200 rpm

	sim.setViewport(Vec(-0.3, 0, 0.3), Vec(0, 0, 0), Vec(0, 0, 1));
	//sim.setViewport(Vec(.4, -0., .4), Vec(0, -0., -0), Vec(0, 0, 1));

	// our plane has a unit normal in the z-direction, with 0 offset.
	//sim.createPlane(Vec(0, 0, 1), 0, 0.5, 0.55);
	sim.createPlane(Vec(0, 0, 1), 0, 0.6, 0.65);


	//double runtime = 120;
	//sim.setBreakpoint(runtime);
	

	sim.start();

	//auto start = std::chrono::steady_clock::now();

	while (sim.RUNNING) {
		std::this_thread::sleep_for(std::chrono::milliseconds(1));
	}

	auto end = std::chrono::steady_clock::now();
	printf("main():Elapsed time:%d ms \n",
		std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

	return 0;
}

